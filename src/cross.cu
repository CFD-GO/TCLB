#include "hip/hip_runtime.h"
#include <stdio.h>
#include "Consts.h"
#include "cross.h"
#include "Global.h"
#include "LatticeContainer.h"
#include <vector>
#include <algorithm>
#include <iostream>
#ifdef CROSS_CPU

uint3 CpuBlock, CpuThread, CpuSize;

void memcpy2D(void * dst_, int dpitch, void * src_, int spitch, int width, int height) {
	char * dst = (char*) dst_, *src = (char*) src_;
	for (int i=0; i<height; i++) {
		memcpy(dst + i*dpitch, src + i*spitch, width);
	}
}

#else

// Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        fprintf(stderr, "[%d] %s in %s at line %d\n", D_MPI_RANK, hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

/*int GetMaxThreads()
{
            hipFuncAttributes * attr = new hipFuncAttributes;
            HANDLE_ERROR( hipFuncGetAttributes(attr, reinterpret_cast<const void*>(RunKernel<Node>)) );
            printf( "Constant mem:%ld\n", attr->constSizeBytes);
            printf( "Local    mem:%ld\n", attr->localSizeBytes);
            printf( "Max  threads:%d\n", attr->maxThreadsPerBlock);
            printf( "Reg   Number:%d\n", attr->numRegs);
            printf( "Shared   mem:%ld\n", attr->sharedSizeBytes);
            return attr->maxThreadsPerBlock;
}
*/
struct ptrpair {
	void ** ptr;
	size_t size;
	ptrpair() { ptr=NULL; size = 0; }
	ptrpair(const ptrpair & p) { ptr=p.ptr; size=p.size; };
	ptrpair(void ** ptr_, size_t size_) { ptr=ptr_; size=size_; };
	inline const bool operator< (const ptrpair & B) const {
		return size < B.size;
	};
};

std::vector< ptrpair > ptrlist;

hipError_t cudaPreAlloc(void ** ptr, size_t size) {
	DEBUG1(printf("Preallocation of %d b\n", (int) size);)
	ptrlist.push_back(ptrpair(ptr, size));
//	return hipMalloc(ptr, size);
	return hipSuccess;
}

#define MEM_ALIGN 128

hipError_t cudaAllocFinalize() {
	sort(ptrlist.begin(), ptrlist.end());
	ptrpair ptr;
	size_t fullsize=0;
	for (int i = 0; i < ptrlist.size(); i++) {
		size_t size = ptrlist[i].size;
		int align = MEM_ALIGN;
		while (align > size) align /= 2;
		size = (((size-1)/align)+1)*align;
		fullsize += size;
		ptrlist[i].size=size;
	}
	char * tmp;
	if (fullsize > 1e9) {
		printf("[%d] Cumulative allocation of %d b (%.1f GB)\n", D_MPI_RANK, (int) fullsize, ((float) fullsize)/1e9);
	} else if (fullsize > 1e6) {
		printf("[%d] Cumulative allocation of %d b (%.1f MB)\n", D_MPI_RANK, (int) fullsize, ((float) fullsize)/1e6);
	} else if (fullsize > 1e3) {
		printf("[%d] Cumulative allocation of %d b (%.1f kB)\n", D_MPI_RANK, (int) fullsize, ((float) fullsize)/1e3);
	} else {
		printf("[%d] Cumulative allocation of %d b\n", D_MPI_RANK, (int) fullsize);
	}
	hipMalloc((void **) &tmp,fullsize);
	if (tmp == NULL) {
		std::cerr << "FATAL ERROR: Not enaught memory! tried to allocate (cumulatice): " << fullsize << " b\n";
		exit(-1);
	}
	CudaMemset( tmp, 0, fullsize );
	while (!ptrlist.empty()) {
		ptr = ptrlist.back();
//		printf("Allocation of %d b\n", (int) ptr.size);
//		hipMalloc(ptr.ptr,ptr.size);
		*(ptr.ptr) = (void **)tmp;
		tmp += ptr.size;
		ptrlist.pop_back();
	}
	return hipSuccess;
}


#endif