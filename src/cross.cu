#include "hip/hip_runtime.h"
#include <stdio.h>
#include "Consts.h"
#include "cross.h"
#include "Global.h"
#include "LatticeContainer.h"
#include <vector>
#include <algorithm>
#include <iostream>

#ifdef CROSS_CPU

uint3 CpuBlock, CpuThread, CpuSize;

void memcpy2D(void * dst_, int dpitch, void * src_, int spitch, int width, int height) {
	char * dst = (char*) dst_, *src = (char*) src_;
	for (int i=0; i<height; i++) {
		memcpy(dst + i*dpitch, src + i*spitch, width);
	}
}

#else

// Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
CudaError HandleError( CudaError err,
                         const char *file,
                         int line ) {
    if (err != CudaSuccess) {
        ERROR("%s in %s at line %d\n", CudaGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    }
	return err;
}

#endif

#ifndef CROSS_SYNCALLOC

        struct ptrpair {
                void ** ptr;
                size_t size;
                ptrpair() { ptr=NULL; size = 0; }
                ptrpair(const ptrpair & p) { ptr=p.ptr; size=p.size; };
                ptrpair(void ** ptr_, size_t size_) { ptr=ptr_; size=size_; };
                inline const bool operator< (const ptrpair & B) const {
                        return size < B.size;
                };
        };

        std::vector< ptrpair > ptrlist;
        std::vector< std::pair< void *, std::vector< ptrpair > > > freelist;

        CudaError cudaPreAlloc(void ** ptr, size_t size) {
                debug1("Preallocation of %d b\n", (int) size);
                ptrlist.push_back(ptrpair(ptr, size));
        //	return hipMalloc(ptr, size);
                return CudaSuccess;
        }

        #define MEM_ALIGN 128

        CudaError cudaAllocFinalize() {
                sort(ptrlist.begin(), ptrlist.end());
                ptrpair ptr;
                size_t fullsize=0;
                for (size_t i = 0; i < ptrlist.size(); i++) {
                        size_t size = ptrlist[i].size;
                        size_t align = MEM_ALIGN;
                        while (align > size) align /= 2;
                        size = (((size-1)/align)+1)*align;
                        fullsize += size;
                        ptrlist[i].size=size;
                }
                char * tmp = NULL;
                if (fullsize > 1e9) {
                        NOTICE("[%d] Cumulative allocation of %d b (%.1f GB)\n", D_MPI_RANK, (int) fullsize, ((float) fullsize)/1e9);
                } else if (fullsize > 1e6) {
                        NOTICE("[%d] Cumulative allocation of %d b (%.1f MB)\n", D_MPI_RANK, (int) fullsize, ((float) fullsize)/1e6);
                } else if (fullsize > 1e3) {
                        NOTICE("[%d] Cumulative allocation of %d b (%.1f kB)\n", D_MPI_RANK, (int) fullsize, ((float) fullsize)/1e3);
                } else {
                        NOTICE("[%d] Cumulative allocation of %d b\n", D_MPI_RANK, (int) fullsize);
                }
                CudaMalloc((void **) &tmp,fullsize);
                if (tmp == NULL) {
                        ERROR("FATAL ERROR: Not enaught memory! tried to allocate (cumulatice): %ld\n", fullsize);
                        exit(-1);
                }
                CudaMemset( tmp, 0, fullsize );
                void * main_ptr = tmp;
                std::vector< ptrpair > tofree;
                while (!ptrlist.empty()) {
                        ptr = ptrlist.back();
                        debug1("[%d] Preallocation gave %d b\n", D_MPI_RANK, (int) ptr.size);
        //		hipMalloc(ptr.ptr,ptr.size);
                        *(ptr.ptr) = (void **)tmp;
                        tmp += ptr.size;
                        tofree.push_back(ptr);
                        ptrlist.pop_back();
                }
                freelist.push_back(std::pair< void *, std::vector< ptrpair > > ( main_ptr, tofree));
                return CudaSuccess;
        }


        CudaError cudaAllocFreeAll() {
                std::pair< void *, std::vector< ptrpair > > ptr_list;
                while (!freelist.empty()) {
                        ptr_list = freelist.back();
                        CudaFree(ptr_list.first);
                        std::vector< ptrpair >::iterator it;
                        for (it=ptr_list.second.begin(); it != ptr_list.second.end(); it++) {
                                *((*it).ptr) = NULL;
                        }
                        freelist.pop_back();
                }
                return CudaSuccess;
        }



#else

        CudaError cudaPreAlloc(void ** ptr, size_t size) {
                debug1("Preallocation of %d b\n", (int) size);
                CudaError ret = CudaMalloc(ptr, size);
                CudaMemset( *ptr, 0, size );
                return ret;
        }

        CudaError cudaAllocFinalize() {
                return CudaSuccess;
        }

        CudaError cudaAllocFreeAll() {
                // TODO: What should go here?? MD
                return CudaSuccess;
        }

#endif

